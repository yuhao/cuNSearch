#include "hip/hip_runtime.h"
#include "cuNSearch.h"
#include "Timing.h"

#include <fstream>
#include <iostream>
#include <vector>
#include <array>
#include <cmath>
#include <limits>
#include <random>
#include <string>

using namespace cuNSearch;

using Real3 = std::array<Real, 3>;
std::vector<Real3> positions;

inline Real3 operator-(const Real3 & left, const Real3 & right)
{
  return Real3{ left[0] - right[0], left[1] - right[1], left[2] - right[2] };
}

//std::size_t const N = 120;
Real const r_omega = static_cast<Real>(0.15);
Real const r_omega2 = r_omega * r_omega;
Real radius;
//Real const radius = static_cast<Real>(2.0) * (static_cast<Real>(2.0) * r_omega / static_cast<Real>(N - 1));

void read_pc_data(const char* data_file) {
  std::ifstream file;

  file.open(data_file);
  if( !file.good() ) {
    std::cerr << "Could not read the frame data...\n";
    //assert(0);
  }

  char line[1024];
  unsigned int lines = 0;

  while (file.getline(line, 1024)) {
    lines++;
  }
  file.clear();
  file.seekg(0, std::ios::beg);
  //float3* points = new float3[lines];
  //*N = lines;

  lines = 0;
  while (file.getline(line, 1024)) {
    Real x, y, z;

    sscanf(line, "%lf,%lf,%lf\n", &x, &y, &z);
    std::array<Real, 3> t = { { static_cast<Real>(x), static_cast<Real>(y), static_cast<Real>(z) } };
    positions.push_back(t);
    //points[lines].x = x;
    //points[lines].y = y;
    //points[lines].z = z;
    //std::cerr << points[lines].x << "," << points[lines].y << "," << points[lines].z << std::endl;
    lines++;
  }

  file.close();

  //return points;
}

void testCuNSearch(const char* data_file)
{
  // read points
  read_pc_data(data_file);
  bool shuffle = false;
  if (shuffle) {
    unsigned seed = std::chrono::system_clock::now()
                        .time_since_epoch()
                        .count();
    std::shuffle(std::begin(positions), std::end(positions), std::default_random_engine(seed));
    //std::cerr << positions[0][0] << ", " << positions[0][1] << ", " << positions[0][2] << std::endl;
  }

  unsigned int numPrims = static_cast<int>(positions.size());
  printf("Number of particles: %d \n", numPrims);

  //Create neighborhood search instance
  NeighborhoodSearch nsearch(radius);
  printf("Radius: %lf \n", radius);

  //Add point set from the test data
  auto pointSetIndex = nsearch.add_point_set(positions.front().data(), positions.size(), true, true);

  for (size_t i = 0; i < 3; i++)
  {
    if (i != 0)
    {
      nsearch.z_sort();
      nsearch.point_set(pointSetIndex).sort_field((Real3*)nsearch.point_set(pointSetIndex).GetPoints());
    }

    Timing::reset();
    Timing::startTiming("Total time");
      nsearch.find_neighbors();
    Timing::stopTiming(true);
    Timing::printAverageTimes();
  }
}

int main(int argc, char* argv[])
{
#ifdef DEBUG
  std::cout << "Debug Build:" << std::endl;

  if(sizeof(Real) == 4)
    std::cout << "Real = float" << std::endl;
  else if (sizeof(Real) == 8)
    std::cout << "Real = double" << std::endl;
#endif

  int device_id = 1;
  hipSetDevice(device_id);
  std::cerr << "\tUsing [" << device_id << "]: " << std::endl;

  std::string outfile;
  outfile = argv[1];
  radius = static_cast<Real>(std::stof(argv[2]));

  testCuNSearch(outfile.c_str());
  std::cout << "Finished Testing" << std::endl;
}
